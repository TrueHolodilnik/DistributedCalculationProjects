#include "hip/hip_runtime.h"
#include "utility.h"
#include <stdio.h>
#include <stdlib.h>
#include <hip/hip_runtime.h>
#include <sys/time.h>

#define SEED 12345678

//Generate random numbers
unsigned int numgen(unsigned int count, unsigned long int dest[])
{

  unsigned int i = 0;

  srandom(SEED);

  while(count--) {
    dest[i++] = random();
  }

  return i;
}

__host__
void errorexit(const char *s) {
    printf("\n%s",s);	
    exit(EXIT_FAILURE);	 	
}

__device__
unsigned long isPrime(unsigned long n)
{
  if (n <= 1)
		return true;
	for (unsigned long int i = 2; i <= sqrtf(n); i++)
		if (n % i == 0)
			return false;
    return true;
}

//Check if number is prime for current index index and increment result if so
__global__
void CheckHowManyPrimes(unsigned long numbers[], unsigned long long *primes, long size)
{
  int index = blockIdx.x * blockDim.x + threadIdx.x;

  if (index < size)
  {
    if (isPrime(numbers[index]))
    {
      atomicAdd(primes, 1);
    }
  }
}

int main(int argc, char **argv)
{

  //Get input
  Args ins__args;
  parseArgs(&ins__args, &argc, argv);
  long inputArgument = ins__args.arg;
  
  //Generate random numbers array
  unsigned long int *numbers = (unsigned long int *)malloc(inputArgument * sizeof(unsigned long int));
  numgen(inputArgument, numbers);

  struct timeval ins__tstart, ins__tstop;
  gettimeofday(&ins__tstart, NULL);


  unsigned long *d_numbers;
  unsigned long long *d_primes;
  unsigned long long primes = 0;
  
  //Move all the necessary data to the GPU mem
  hipMalloc(&d_numbers, inputArgument * sizeof(unsigned long));
  hipMalloc(&d_primes, sizeof(unsigned long long));
  hipMemcpy(d_numbers, numbers, inputArgument * sizeof(unsigned long), hipMemcpyHostToDevice);
  hipMemcpy(d_primes, &primes, sizeof(unsigned long long), hipMemcpyHostToDevice);
  
  //Run main calculations on GPU, grid and block size definitions
  int blockSize = 256;
  int gridSize = (inputArgument + blockSize - 1) / blockSize;
  CheckHowManyPrimes<<<gridSize, blockSize>>>(d_numbers, d_primes, inputArgument);
  
  //Get the result
  hipMemcpy(&primes, d_primes, sizeof(unsigned long long), hipMemcpyDeviceToHost);
  printf("Number of primes: %lld\n", primes);
	
  //Clear used GPU mem
  hipFree(d_numbers);
  hipFree(d_primes);

  free(numbers);

  //Synchronize CUDA computations
  gettimeofday(&ins__tstop, NULL);
  
  //Show elapsed time
  ins__printtime(&ins__tstart, &ins__tstop, ins__args.marker);
  
}
